#include "hip/hip_runtime.h"
//Compile with: nvcc CUDAfft2.0.cu -I/home/phyd57/N_Body1/9.2/include -L/home/phyd57/N_Body1/9.2/lib64 -lcufft -o CUDAfftcu2.out -I/usr/local/dislin -ldislin
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>

#include <stdlib.h>
#include<stdio.h>
#include <iostream>
#include<time.h>

#include "dislin.h"

#define N 256 // N is the sidelength of the image -> N^3 pixels in entire image
#define M 100000000 //M is the number of particles.
#define block_size_x 2 
#define block_size_y 2
#define block_size_z 2

float den_array[N][N][N];
float grav_po[N][N][N];
//float particleArray[M][7];
float image[N/2][N/2];

__global__ void real2complex(hipfftComplex *c, float *a, int n);
__global__ void complex2real_scaled(float *a, hipfftComplex *c, float scale, int n);
__global__ void solve_poisson(hipfftComplex *c, float *k_xyz, int n);


void FFT_poisson(float den_array[N][N][N], float grav_po[N][N][N])
{
	int x, y, z, i;

	float *k_xyz, *den;
	k_xyz = (float *)malloc(sizeof(float)*N);
	den = (float *)malloc(sizeof(float)*N*N*N);

	float *k_xyz_d, *den_d;
	hipfftComplex *den_complex_d;
	hipMalloc((void **)&k_xyz_d, sizeof(float) * N);
	hipMalloc((void **)&den_d, sizeof(float) * N * N * N);
	hipMalloc((void **)&den_complex_d, sizeof(hipfftComplex) * N * N * N);

	#pragma omp for
	for (x = 0; x < N; x++)
		for (y = 0; y < N; y++)
			for (z = 0; z < N; z++)
				den[x + y*N + z*N*N] = 4.0 * 3.14159 * 1.4006 * den_array[x][y][z];
				//Where 1.4006 is G in units kPc**3/solar_mass * 10kyears

	float* den_inital = (float *)malloc(sizeof(float) * N * N * N);

	#pragma omp for
	for (i = 0; i < N * N; i++)
		den_inital[i] = den[i];

	#pragma omp for
	for (i = 0; i < N; i++)
	{
		if (i < N/2)
		{
			k_xyz[i] = i;
		}
		else
		{
			k_xyz[i] = i-N;
		}
	}

	hipMemcpy(k_xyz_d, k_xyz, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(den_d, den, sizeof(float)*N*N*N, hipMemcpyHostToDevice);

	hipfftHandle plan;
	hipfftPlan3d(&plan,N,N,N,HIPFFT_C2C);

	/* Compute the execution configuration, block_size_x*block_size_y*block_size_z = number of threads */
	dim3 dimBlock(block_size_x, block_size_y, block_size_z);
	dim3 dimGrid(N/dimBlock.x, N/dimBlock.y, N/dimBlock.z);
	/* Handle N not multiple of block_size_x, block_size_y, or block_size_y */
	if (N % block_size_x != 0) dimGrid.x += 1;
	if (N % block_size_y != 0) dimGrid.y += 1;
	if (N % block_size_z != 0) dimGrid.z += 1;

	real2complex<<<dimGrid, dimBlock>>>(den_complex_d, den_d, N);

	hipfftExecC2C(plan, den_complex_d, den_complex_d, HIPFFT_FORWARD);

	solve_poisson<<<dimGrid, dimBlock>>>(den_complex_d, k_xyz_d, N);

	hipfftExecC2C(plan, den_complex_d, den_complex_d, HIPFFT_BACKWARD);

	float scale = 1.0f / (N*N*N);
	complex2real_scaled<<<dimGrid, dimBlock>>>(den_d, den_complex_d, scale, N);
	

	hipMemcpy(den, den_d, sizeof(float)*N*N*N, hipMemcpyDeviceToHost);

	#pragma omp for
	for (x = 0; x < N; x++)
		for (y = 0; y < N; y++)
			for (z = 0; z < N; z++)
				grav_po[x][y][z] = den[x + y*N + z*N*N];

	/* Destroy plan and clean up memory on device*/
	hipFree(k_xyz);
	hipFree(den);
	hipFree(den_inital);
	hipfftDestroy(plan);
	hipFree(den_complex_d);
	hipFree(den);
	hipFree(k_xyz_d);
}


void make_image(float array[N][N][N], const char *output_name, const char *title)
{
	int x, y, z;
	float Max = -500.0, Min = 500.0;
	
	#pragma omp for
	for (x = 0; x < N/2; x++)
		for (y = 0; y < N/2; y++)
			image[x][y] = 0.0;

	#pragma omp for
	for (x = 0; x < N/2; x++)
		for (y = 0; y < N/2; y++)
			for (z = 0; z < N/2; z++)
				image[x][y] += array[x+N/4][y+N/4][z+N/4];

	#pragma omp for
	for (x = 0; x < N/2; x++)
	{
		for (y = 0; y < N/2; y++)
		{
			if (image[x][y] > Max)
			{
				Max = image[x][y];
			}
			if (image[x][y] < Min)
			{
				Min = image[x][y];
			}
		}
	}

	metafl("PNG");
	setfil(output_name);
	//metafl("CONS");
	disini();
	pagera();
	hwfont();

	titlin(title, 4);
	//titlin("anthing below", 2)

	name("X [kP]", "x");
	name("Y [kP]", "y");
	name("Potential in Z", "z");

	intax()	;
	autres(N/2,N/2);
	axspos(300,1850);
	ax3len(1600,1600,1600);
	
	labdig(6, "Z");
	graf3(-N/4, N/4, -N/4, N/40, -N/4, N/4, -N/4, N/40, Min, Max, Min, (Max-Min)/10);
	crvmat((float *)image, N/2, N/2 , 1, 1);

	height(50);
	title();
	disfin();
}

void *CM_finder(int galaxy_ID, float xyz_array[3])
{
	/*
	Fills xyz_array with the z, y, and z values of the CM of
	a given galaxy, in that order.
	galaxy_ID is 1 for galaxy 1 and 2 for galaxy 2.
	*/
	int i, n;
	
	if (galaxy_ID == 1) n = 0;
	else n = M/2;
	
	#pragma omp for
	for (i = 0; i < M/2; i ++)
	{
		xyz_array[0] = part_array[i+n][0];
		xyz_array[1] = part_array[i+n][1];
		xyz_array[2] = part_array[i+n][2];
	}
	
	xyz_array[0] /= (float)M/2;
	xyz_array[1] /= (float)M/2;
	xyz_array[2] /= (float)M/2;
}
/*
void initial_velocity(int galaxy_ID)
{
	float CM_array[3];
	CM_finder(galaxy_ID, xyz_array[3]);
	
	int i, n;
	float x, y, z, r, v;
	
	if (galaxy_ID == 1) n = 0;
	else n = M/2;
	
	#pragma omp for
	for (i = 0; i < M/2; i ++)
	{
		x = xyz_array[0] - part_array[i+n][0];
		y = xyz_array[1] - part_array[i+n][1];
		z = xyz_array[2] - part_array[i+n][2];
		r = x*x + y*y + z*z;
		r = pow(r, 0.5)
		
		v = //pow(G*m*M/r, 0.5); need the unit of time to know the value of G
		
		//from there I need the direction it moves from there.
	}
	
	pow(value, 0.5);
	
	//Also should add the 402000 km/h here
}*/

void densArray(float **particleArray, float*** threedArray) {
	int i, j, k = 0;
    // dynamically allocate memory of size M*N*O
	// assign values to allocated memory
	/*
	for (i = 0; i < I; i++) {
		threedArray[i] = (float**)malloc(J * sizeof(float*));
        if (threedArray[i] == NULL) {
			fprintf(stderr, "Out of memory");
			exit(0);
		}
        for (j = 0; j < J; j++) {
			threedArray[i][j] = (float*)malloc(K * sizeof(float));
            if (threedArray[i][j] == NULL) {
				fprintf(stderr, "Out of memory");
				exit(0);
			}
	}*/

    // assign values to allocated memory
	#pragma omp for
	for (i=0; i < M; i++) {
        // printf("%d\n", i);
        // printf("%d\n", (int)floorf(particleArray[i][0]));
        // printf("%d %d %d \n", (int)floorf(particleArray[i][0]), (int)floorf(particleArray[i][1]), (int)floorf(particleArray[i][2]));
        threedArray[(int)floorf(particleArray[i][0])][(int)floorf(particleArray[i][1])][(int)floorf(particleArray[i][2])] =
        threedArray[(int)floorf(particleArray[i][0])][(int)floorf(particleArray[i][1])][(int)floorf(particleArray[i][2])] + 1;
	}
    // // print the 3D array
	// for (i = 0; i < I; i++)
	// {
	// 	for (j = 0; j < J; j++)
	// 	{
	// 		for (k = 0; k < K; k++)
	// 			printf("%f\n", threedArray[i][j][k]);
	//    	}
	// }
}

void center_diff(int xN, int yN, int zN, float*** grav_po, float **particleArray) {
    int i, j, k, l;
    float v_half, x, v;

    // float gx[I][J][K], gy[I][J][K], gz[I][J][K];
    // float (*g)[I][J][K];

    // for(i=1; i<xN; i++){
    //     for(j=1; j<yN-1; j++){
    //         for(k=1; k<zN-1; k++){
    //             gx[i][j][k] = (grav_po[i+1][j][k] - grav_po[i-1][j][k])/(2); // get g for each directions
    //             gy[i][j][k] = (grav_po[i][j+1][k] - grav_po[i][j-1][k])/(2);
    //             gz[i][j][k] = (grav_po[i][j][k+1] - grav_po[i][j][k-1])/(2);
    //         }
    //     }
    // }
    // printf("g force created\n");

	#pragma omp for
    for(i=0; i<M; i++){
        for(l=0; l<1; l++){
            v_half = particleArray[i][l+3] + 
            (grav_po[(int)round(particleArray[i][0])+1][(int)round(particleArray[i][1])][(int)round(particleArray[i][2])]
                - grav_po[(int)round(particleArray[i][0])-1][(int)round(particleArray[i][1])][(int)round(particleArray[i][2])])/(2);
            x = particleArray[i][l] + v_half;
            v = v_half + 
            (grav_po[(int)round(particleArray[i][0])+1][(int)round(particleArray[i][1])][(int)round(particleArray[i][2])]
                - grav_po[(int)round(particleArray[i][0])-1][(int)round(particleArray[i][1])][(int)round(particleArray[i][2])])/(2);
            particleArray[i][l+3] = v;
        }
        for(l=1; l<2; l++){
            v_half = particleArray[i][l+3] + 
            (grav_po[(int)round(particleArray[i][0])][(int)round(particleArray[i][1])+1][(int)round(particleArray[i][2])] 
            - grav_po[(int)round(particleArray[i][0])][(int)round(particleArray[i][1])-1][(int)round(particleArray[i][2])])/(2);
            x = particleArray[i][l] + v_half;
            v = v_half + 
            (grav_po[(int)round(particleArray[i][0])][(int)round(particleArray[i][1])+1][(int)round(particleArray[i][2])] 
            - grav_po[(int)round(particleArray[i][0])][(int)round(particleArray[i][1])-1][(int)round(particleArray[i][2])])/(2);
            particleArray[i][l+3] = v;
        }
        for(l=2; l<3; l++){
            v_half = particleArray[i][l+3] + 
            (grav_po[(int)round(particleArray[i][0])][(int)round(particleArray[i][1])][(int)round(particleArray[i][2])+1] 
            - grav_po[(int)round(particleArray[i][0])][(int)round(particleArray[i][1])][(int)round(particleArray[i][2])-1])/(2);
            x = particleArray[i][l] + v_half;
            v = v_half + 
            (grav_po[(int)round(particleArray[i][0])][(int)round(particleArray[i][1])][(int)round(particleArray[i][2])+1] 
            - grav_po[(int)round(particleArray[i][0])][(int)round(particleArray[i][1])][(int)round(particleArray[i][2])-1])/(2);
            particleArray[i][l+3] = v;
        }
            // move all particles
            // updater(particleArray[i][l+3], particleArray[i][l],
            // *g[(int)round(particleArray[i][0])][(int)round(particleArray[i][1])][(int)round(particleArray[i][2])]);
            // v_half = particleArray[i][l+3] +
            // *g[(int)round(particleArray[i][0])][(int)round(particleArray[i][1])][(int)round(particleArray[i][2])]/2+
            // 1190/(pow(particleArray[i][0],2)+pow(particleArray[i][1],2)+pow(particleArray[i][2],2));
            // x = particleArray[i][l] + v_half;
            // v = v_half +
            // *g[(int)round(particleArray[i][0])][(int)round(particleArray[i][1])][(int)round(particleArray[i][2])]/2;
            // particleArray[i][l] = x;
            // particleArray[i][l+3] = v;
    }

    // // update density array (TDB)
    // printf("density array updater initiated\n");
}

int main()
{
	//initialize particle array without velocity.
	int i, j, k, index, max_number, min_number, counter;
	float t, dt, X, Y, R, V;
	float *particleArray = (float *)malloc(M * sizeof(float *));
	
	t = 0.0;
	dt = 1.0;

	#pragma omp for
	for (i = 0; i < M; i++) {
		particleArray[i] = (float *)malloc(M * sizeof(float*));

		if (particleArray[i] == NULL) {
			fprintf(stderr, "Out of memory");
			exit(0);
		}
	}

	// first galaxy population
	#pragma omp for
    for (i = 0; i < (int)(M*0.05/2); i++) {
        particleArray[i][0] = 2*1.41*cos((float)(rand()%629)/100) + 96.0;
        particleArray[i][1] = 2*1.41*sin((float)(rand()%629)/100) + 96.0;
        particleArray[i][2] = (float)(rand()%(50+1))/1000 + 128.0; // rand() % (max_number + 1 - minimum_number) + minimum_number
        X = particleArray[i][0] - 96;
        Y = particleArray[i][1] - 96;
        R = sqrt(pow(X,2) + pow(Y,2));
        V = sqrt(1190.0*R);
        particleArray[i][3] = Y/R*V + 0.04;
        particleArray[i][4] = X/R*V + 0.04;
        particleArray[i][5] = 0;
    }
	
	#pragma omp for
    for (index=1; index<11; index++){
        for (i = (int)(M*0.05/2+((index-1)*0.095*M/2)); i < (int)(M*0.05/2+((index)*0.095*M/2)); i++) {
            particleArray[i][0] = (2+index)*1.41*cos((float)(rand()%629)/100) + 96.0;
            particleArray[i][1] = (2+index)*1.41*sin((float)(rand()%629)/100) + 96.0;
            particleArray[i][2] = (float)(rand()%(50+1))/1000 + 128.0;
            X = particleArray[i][0] - 96;
            Y = particleArray[i][1] - 96;
            R = sqrt(pow(X,2) + pow(Y,2));
            V = sqrt(1190.0*R);
            particleArray[i][3] = Y/R*V + 0.04;
            particleArray[i][4] = X/R*V + 0.04;
			particleArray[i][5] = 0;
        }
    }
	
	#pragma omp for
	for (i = 0; i < (int)(M/2); i++) {
		for (j=6;j<7;j++){
			particleArray[i][j] = 0.0; // 0.0 is indicator for Milky Way
		}
	}

	// second galaxy population
	#pragma omp for
	for (i = (int)(M*0.05/2+((10)*0.095*M/2)); i < (int)(M*0.05/2+((10)*0.095*M/2))+(int)(M*0.05/2); i++) {
        particleArray[i][0] = 2*1.41*cos((float)(rand()%629)/100)  + 160.0;
        particleArray[i][1] = 2*1.41*sin((float)(rand()%629)/100)  + 160.0;
        particleArray[i][2] = (float)(rand()%(50+1))/1000 + 128.0;
        X = particleArray[i][0] - 96;
        Y = particleArray[i][1] - 96;
        R = sqrt(pow(X,2) + pow(Y,2));
        V = sqrt(1190.0*R);
        particleArray[i][3] = Y/R*V - 0.04;
        particleArray[i][4] = X/R*V - 0.04;
        particleArray[i][5] = 0;
    }

	#pragma omp for
	for (index=11; index<21; index++){
        for (i = (int)(M*0.05+((index-1)*0.095*M/2)); i < (int)(M*0.05+((index)*0.095*M/2)); i++) {
            particleArray[i][0] = (2+index-10)*1.41*cos((float)(rand()%629)/100)  + 160.0;
            particleArray[i][1] = (2+index-10)*1.41*sin((float)(rand()%629)/100)  + 160.0;
            particleArray[i][2] = (float)(rand()%(150+1))/1000 + 128.0;
            X = particleArray[i][0] - 96;
            Y = particleArray[i][1] - 96;
            R = sqrt(pow(X,2) + pow(Y,2));
            V = sqrt(1190*R);
            particleArray[i][3] = Y/R*V - 0.04;
            particleArray[i][4] = X/R*V - 0.04;
            particleArray[i][5] = 0;
        }
    }

	#pragma omp for
	for (i = (int)(M/2); i < M; i++) {
		particleArray[i][6] = 1.0; // 1.0 is indicator for Andromeda
	}
	
	//create initial velocity, for each array.
	
	///Repeat until finished.
	while (t < 500)
	{
		densArray(particleArray, den_array);
		FFT_poisson(den_array, grav_po);
		enter_diff(256, 256, 256, grav_po, particleArray);
		
		if (time == 0.0)
		{
			make_image(den_array, "Initial.png", "Initial density of the system");
		}
		
		if (time == 125.0)
		{
			make_image(den_array, "fourth.png", "Density of the system after 1,250,000 years");
		}
		
		if (time == 250.0)
		{
			make_image(den_array, "half.png", "Density of the system after 2,500,000 years");
		}
		
		if (time == 375.0)
		{
			make_image(den_array, "three_fourths.png", "Density of the system after 3,750,000 years");
		}
		
		time += dt;
	}
	
	//Fill density array with both galaxies
	//Find potential
	//update particle with potential
	
	//end.
	
	make_image(den_array, "final.png", "Density of the system after 5,000,000 years");
	
	return 0;

}


__global__ void real2complex(hipfftComplex *c, float *a, int n)
{
    /* compute idx, idy, and idz, the location of the element in the original NxNxN array */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < n && idy < n && idz < n)
	{
		int index = idx + idy*n + idz*n*n;
		c[index].x = a[index];
		c[index].y = 0.0f;
	}
}

__global__ void complex2real_scaled(float *a, hipfftComplex *c, float scale, int n)
{
	/* compute idx and idy, the location of the element in the original NxN array */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < n && idy < n && idz < n)
	{
		int index = idx + idy*n + idz*n*n;
		a[index] = scale * c[index].x;
	}
}


__global__ void solve_poisson(hipfftComplex *c, float *k_xyz, int n)
{
	/* compute idx and idy, the location of the element in the original NxN array */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < n && idy < n && idz < n)
	{
		int index = idx + idy*n + idz*n*n;
		float scale = -(k_xyz[idx]*k_xyz[idx] + k_xyz[idy]*k_xyz[idy] + k_xyz[idz]*k_xyz[idz]) + 0.00001f;
		if (idx == 0 && idy == 0 && idz == 0) scale = 1.0f;
		scale = 1.0f / scale;
		c[index].x *= scale;
		c[index].y *= scale;
	}
}