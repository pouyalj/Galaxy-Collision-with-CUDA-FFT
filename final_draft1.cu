#include "hip/hip_runtime.h"
//Compile with: nvcc CUDAfft2.0.cu -I/home/phyd57/N_Body1/9.2/include -L/home/phyd57/N_Body1/9.2/lib64 -lcufft -o CUDAfftcu2.out -I/usr/local/dislin -ldislin
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>

#include <stdlib.h>
#include<stdio.h>
#include <iostream>

#include "dislin.h"

#define N 256 // N is the sidelength of the image -> N^3 pixels in entire image
#define M 100000000 //M is the number of particles.
#define block_size_x 2 
#define block_size_y 2
#define block_size_z 2

float den_array[N][N][N];
float grav_po[N][N][N];
float part_array[M][6];
float image[N/2][N/2];

__global__ void real2complex(hipfftComplex *c, float *a, int n);
__global__ void complex2real_scaled(float *a, hipfftComplex *c, float scale, int n);
__global__ void solve_poisson(hipfftComplex *c, float *k_xyz, int n);


void FFT_poisson(float den_array[N][N][N], float grav_po[N][N][N])
{
	int x, y, z, i;

	float *k_xyz, *den;
	k_xyz = (float *)malloc(sizeof(float)*N);
	den = (float *)malloc(sizeof(float)*N*N*N);

	float *k_xyz_d, *den_d;
	hipfftComplex *den_complex_d;
	hipMalloc((void **)&k_xyz_d, sizeof(float) * N);
	hipMalloc((void **)&den_d, sizeof(float) * N * N * N);
	hipMalloc((void **)&den_complex_d, sizeof(hipfftComplex) * N * N * N);

	#pragma omp for
	for (x = 0; x < N; x++)
		for (y = 0; y < N; y++)
			for (z = 0; z < N; z++)
				den[x + y*N + z*N*N] = den_array[x][y][z];

	float* den_inital = (float *)malloc(sizeof(float) * N * N * N);
	for (i = 0; i < N * N; i++)
		den_inital[i] = den[i];

	for (i = 0; i < N; i++)
	{
		if (i < N/2)
		{
			k_xyz[i] = i;
		}
		else
		{
			k_xyz[i] = i-N;
		}
	}

	hipMemcpy(k_xyz_d, k_xyz, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(den_d, den, sizeof(float)*N*N*N, hipMemcpyHostToDevice);

	hipfftHandle plan;
	hipfftPlan3d(&plan,N,N,N,HIPFFT_C2C);

	/* Compute the execution configuration, block_size_x*block_size_y*block_size_z = number of threads */
	dim3 dimBlock(block_size_x, block_size_y, block_size_z);
	dim3 dimGrid(N/dimBlock.x, N/dimBlock.y, N/dimBlock.z);
	/* Handle N not multiple of block_size_x, block_size_y, or block_size_y */
	if (N % block_size_x != 0) dimGrid.x += 1;
	if (N % block_size_y != 0) dimGrid.y += 1;
	if (N % block_size_z != 0) dimGrid.z += 1;

	real2complex<<<dimGrid, dimBlock>>>(den_complex_d, den_d, N);

	hipfftExecC2C(plan, den_complex_d, den_complex_d, HIPFFT_FORWARD);

	solve_poisson<<<dimGrid, dimBlock>>>(den_complex_d, k_xyz_d, N);

	hipfftExecC2C(plan, den_complex_d, den_complex_d, HIPFFT_BACKWARD);

	float scale = 1.0f / (N*N*N);
	complex2real_scaled<<<dimGrid, dimBlock>>>(den_d, den_complex_d, scale, N);
	

	hipMemcpy(den, den_d, sizeof(float)*N*N*N, hipMemcpyDeviceToHost);

	#pragma omp for
	for (x = 0; x < N; x++)
		for (y = 0; y < N; y++)
			for (z = 0; z < N; z++)
				grav_po[x][y][z] = den[x + y*N + z*N*N];

	/* Destroy plan and clean up memory on device*/
	hipFree(k_xyz);
	hipFree(den);
	hipFree(den_inital);
	hipfftDestroy(plan);
	hipFree(den_complex_d);
	hipFree(den);
	hipFree(k_xyz_d);
}


void make_image(float array[N][N][N], const char *output_name)
{
	int x, y, z;
	float Max = -500.0, Min = 500.0;
	
	#pragma omp for
	for (x = 0; x < N/2; x++)
		for (y = 0; y < N/2; y++)
			image[x][y] = 0.0;

	#pragma omp for
	for (x = 0; x < N/2; x++)
		for (y = 0; y < N/2; y++)
			for (z = 0; z < N/2; z++)
				image[x][y] += array[x+N/4][y+N/4][z+N/4];

	#pragma omp for
	for (x = 0; x < N/2; x++)
	{
		for (y = 0; y < N/2; y++)
		{
			if (image[x][y] > Max)
			{
				Max = image[x][y];
			}
			if (image[x][y] < Min)
			{
				Min = image[x][y];
			}
		}
	}

	metafl("PNG");
	setfil(output_name);
	//metafl("CONS");
	disini();
	pagera();
	hwfont();

	titlin("Potential map", 4);
	//titlin("anthing below", 2)

	name("X [kP]", "x");
	name("Y [kP]", "y");
	name("Potential in Z", "z");

	intax()	;
	autres(N/2,N/2);
	axspos(300,1850);
	ax3len(1600,1600,1600);
	
	labdig(6, "Z");
	graf3(-N/4, N/4, -N/4, N/40, -N/4, N/4, -N/4, N/40, Min, Max, Min, (Max-Min)/10);
	crvmat((float *)image, N/2, N/2 , 1, 1);

	height(50);
	title();
	disfin();
}

void *CM_finder(int galaxy_ID, float xyz_array[3])
{
	/*
	Fills xyz_array with the z, y, and z values of the CM of
	a given galaxy, in that order.
	galaxy_ID is 1 for galaxy 1 and 2 for galaxy 2.
	*/
	int i, n;
	
	if (galaxy_ID == 1) n = 0;
	else n = M/2;
	
	#pragma omp for
	for (i = 0; i < M/2; i ++)
	{
		xyz_array[0] = part_array[i+n][0];
		xyz_array[1] = part_array[i+n][1];
		xyz_array[2] = part_array[i+n][2];
	}
	
	xyz_array[0] /= (float)M/2;
	xyz_array[1] /= (float)M/2;
	xyz_array[2] /= (float)M/2;
}

void initial_velocity(int galaxy_ID)
{
	float CM_array[3];
	CM_finder(galaxy_ID, xyz_array[3]);
	
	int i, n;
	float x, y, z, r, v;
	
	if (galaxy_ID == 1) n = 0;
	else n = M/2;
	
	#pragma omp for
	for (i = 0; i < M/2; i ++)
	{
		x = xyz_array[0] - part_array[i+n][0];
		y = xyz_array[1] - part_array[i+n][1];
		z = xyz_array[2] - part_array[i+n][2];
		r = x*x + y*y + z*z;
		r = pow(r, 0.5)
		
		v = //pow(G*m*M/r, 0.5); need the unit of time to know the value of G
		
		//from there I need the direction it moves from there.
	}
	
	pow(value, 0.5);
	
	//Also should add the 402000 km/h here
}

int main()
{
	int i, j, k;

	//initialize particle array without velocity.
	
	//make density array of G1.
	//call to find potential.
	//use potential to find initial velocity.
	//update particle array.
	
	//make density array of G2
	//call to find potential.
	//use potential to find initial velocity.
	//update particle array.
	
	///Repeat until finished.
	//Fill density array with both galaxies
	//Find potential
	//update particle with potential
	
	//end.
	
	return 0;

}


__global__ void real2complex(hipfftComplex *c, float *a, int n)
{
    /* compute idx, idy, and idz, the location of the element in the original NxNxN array */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < n && idy < n && idz < n)
	{
		int index = idx + idy*n + idz*n*n;
		c[index].x = a[index];
		c[index].y = 0.0f;
	}
}

__global__ void complex2real_scaled(float *a, hipfftComplex *c, float scale, int n)
{
	/* compute idx and idy, the location of the element in the original NxN array */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < n && idy < n && idz < n)
	{
		int index = idx + idy*n + idz*n*n;
		a[index] = scale * c[index].x;
	}
}


__global__ void solve_poisson(hipfftComplex *c, float *k_xyz, int n)
{
	/* compute idx and idy, the location of the element in the original NxN array */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < n && idy < n && idz < n)
	{
		int index = idx + idy*n + idz*n*n;
		float scale = -(k_xyz[idx]*k_xyz[idx] + k_xyz[idy]*k_xyz[idy] + k_xyz[idz]*k_xyz[idz]) + 0.00001f;
		if (idx == 0 && idy == 0 && idz == 0) scale = 1.0f;
		scale = 1.0f / scale;
		c[index].x *= scale;
		c[index].y *= scale;
	}
}