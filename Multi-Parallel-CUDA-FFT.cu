#include "hip/hip_runtime.h"
//Compile with: nvcc final_draft1.cu -Xcompiler -fopenmp  -I/home/phyd57/N_Body1/9.2/include -L/home/phyd57/N_Body1/9.2/lib64 -lcufft -o CUDAfftcu2.out -I/usr/local/dislin -ldislin
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include<stdio.h>
#include <iostream>
#include<time.h>

#include "dislin.h"

#define N 256 // N is the sidelength of the image -> N^3 pixels in entire image
#define M 100000000 //M is the number of particles.
#define block_size_x 2 
#define block_size_y 2
#define block_size_z 2

float den_array[N][N][N];
float grav_po[N][N][N];
//float particleArray[M][7];
float image[N/2][N/2];
float G = 1.139430*pow(10,-17) * 840;

__global__ void real2complex(hipfftComplex *c, float *a, int n);
__global__ void complex2real_scaled(float *a, hipfftComplex *c, float scale, int n);
__global__ void solve_poisson(hipfftComplex *c, float *k_xyz, int n);


void FFT_poisson(float*** den_array, float grav_po[N][N][N])
{
	int x, y, z, i;

	float *k_xyz, *den;
	k_xyz = (float *)malloc(sizeof(float)*N);
	den = (float *)malloc(sizeof(float)*N*N*N);

	float *k_xyz_d, *den_d;
	hipfftComplex *den_complex_d;
	hipMalloc((void **)&k_xyz_d, sizeof(float) * N);
	hipMalloc((void **)&den_d, sizeof(float) * N * N * N);
	hipMalloc((void **)&den_complex_d, sizeof(hipfftComplex) * N * N * N);

	#pragma omp for
	for (x = 0; x < N; x++)
		for (y = 0; y < N; y++)
			for (z = 0; z < N; z++)
				// den[x + y*N + z*N*N] = 4.0 * 3.14159 * 4.9*pow(10,-14) * den_array[x][y][z];
				den[x + y*N + z*N*N] = 4.0 * 3.14159 * 1.139430*pow(10,-17) * den_array[x][y][z] * 840;
				//Where 1.4006 is G in units kPc**3/solar_mass * 10kyears

	float* den_inital = (float *)malloc(sizeof(float) * N * N * N);

	#pragma omp for
	for (i = 0; i < N * N; i++)
		den_inital[i] = den[i];

	#pragma omp for
	for (i = 0; i < N; i++)
	{
		if (i < N/2)
		{
			k_xyz[i] = i;
		}
		else
		{
			k_xyz[i] = i-N;
		}
	}

	hipMemcpy(k_xyz_d, k_xyz, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(den_d, den, sizeof(float)*N*N*N, hipMemcpyHostToDevice);

	hipfftHandle plan;
	hipfftPlan3d(&plan,N,N,N,HIPFFT_C2C);

	/* Compute the execution configuration, block_size_x*block_size_y*block_size_z = number of threads */
	dim3 dimBlock(block_size_x, block_size_y, block_size_z);
	dim3 dimGrid(N/dimBlock.x, N/dimBlock.y, N/dimBlock.z);
	/* Handle N not multiple of block_size_x, block_size_y, or block_size_y */
	if (N % block_size_x != 0) dimGrid.x += 1;
	if (N % block_size_y != 0) dimGrid.y += 1;
	if (N % block_size_z != 0) dimGrid.z += 1;

	real2complex<<<dimGrid, dimBlock>>>(den_complex_d, den_d, N);

	hipfftExecC2C(plan, den_complex_d, den_complex_d, HIPFFT_FORWARD);

	solve_poisson<<<dimGrid, dimBlock>>>(den_complex_d, k_xyz_d, N);

	hipfftExecC2C(plan, den_complex_d, den_complex_d, HIPFFT_BACKWARD);

	float scale = 1.0f / (N*N*N);
	complex2real_scaled<<<dimGrid, dimBlock>>>(den_d, den_complex_d, scale, N);
	

	hipMemcpy(den, den_d, sizeof(float)*N*N*N, hipMemcpyDeviceToHost);

	#pragma omp for
	for (x = 0; x < N; x++)
		for (y = 0; y < N; y++)
			for (z = 0; z < N; z++)
				grav_po[x][y][z] = den[x + y*N + z*N*N];

	/* Destroy plan and clean up memory on device*/
	hipFree(k_xyz);
	hipFree(den);
	hipFree(den_inital);
	hipfftDestroy(plan);
	hipFree(den_complex_d);
	hipFree(den);
	hipFree(k_xyz_d);
}

void make_image(float*** array, const char *output_name, const char *title0)
{
	int x, y, z;
	float Max = -500.0, Min = 500.0;
	
	#pragma omp for
	for (x = 0; x < N/2; x++)
		for (y = 0; y < N/2; y++)
			image[x][y] = 0.0;

	#pragma omp for
	for (x = 0; x < N/2; x++)
		for (y = 0; y < N/2; y++)
			for (z = 0; z < N/2; z++)
				image[x][y] += array[x+N/4][y+N/4][z+N/4];

	#pragma omp for
	for (x = 0; x < N/2; x++)
	{
		for (y = 0; y < N/2; y++)
		{
			if (image[x][y] > Max)
			{
				Max = image[x][y];
			}
			if (image[x][y] < Min)
			{
				Min = image[x][y];
			}
		}
	}

	metafl("PNG");
	setfil(output_name);
	//metafl("CONS");
	disini();
	pagera();
	hwfont();

	titlin(title0, 4);
	//titlin("anthing below", 2)

	name("X [kP]", "x");
	name("Y [kP]", "y");
	name("Potential in Z", "z");

	intax()	;
	autres(N/2,N/2);
	axspos(300,1850);
	ax3len(1600,1600,1600);
	
	labdig(6, "Z");
	graf3(-N/4, N/4, -N/4, N/40, -N/4, N/4, -N/4, N/40, Min, Max, Min, (Max-Min)/10);
	crvmat((float *)image, N/2, N/2 , 1, 1);

	height(50);
	title();
	disfin();
}

void CM_finder(int galaxy_ID, float xyz_array[3], float **part_array)
{
	/*
	Fills xyz_array with the x, y, and z values of the CM of
	a given galaxy, in that order.
	galaxy_ID is 1 for galaxy 1 and 2 for galaxy 2.
	*/
	int i, n;
	
	if (galaxy_ID == 1) n = 0;
	else n = M/2;
	
	#pragma omp for
	for (i = 0; i < M/2; i ++)
	{
		xyz_array[0] += part_array[i+n][0];
		xyz_array[1] += part_array[i+n][1];
		xyz_array[2] += part_array[i+n][2];
	}
	
	xyz_array[0] /= (float)M/2;
	xyz_array[1] /= (float)M/2;
	xyz_array[2] /= (float)M/2;

}
/*
void initial_velocity(int galaxy_ID)
{
	float CM_array[3];
	CM_finder(galaxy_ID, xyz_array[3]);
	
	int i, n;
	float x, y, z, r, v;
	
	if (galaxy_ID == 1) n = 0;
	else n = M/2;
	
	#pragma omp for
	for (i = 0; i < M/2; i ++)
	{
		x = xyz_array[0] - part_array[i+n][0];
		y = xyz_array[1] - part_array[i+n][1];
		z = xyz_array[2] - part_array[i+n][2];
		r = x*x + y*y + z*z;
		r = pow(r, 0.5)
		
		v = //pow(G*m*M/r, 0.5); need the unit of time to know the value of G
		
		//from there I need the direction it moves from there.
	}
	
	pow(value, 0.5);
	
	//Also should add the 402000 km/h here
}*/

void densArray(float **particleArray, float*** den_array) {
	int i = 0;
	int j = 0;
	int k = 0;
    // dynamically allocate memory of size M*N*O
	// assign values to allocated memory
	// for (i = 0; i < N; i++) {
	// 	den_array[i] = (float**)malloc(N * sizeof(float*));
    //     if (den_array[i] == NULL) {
	// 		fprintf(stderr, "Out of memory");
	// 		exit(0);
	// 	}
    //     for (j = 0; j < N; j++) {
	// 		den_array[i][j] = (float*)malloc(N * sizeof(float));
    //         if (den_array[i][j] == NULL) {
	// 			fprintf(stderr, "Out of memory");
	// 			exit(0);
	// 		}
	// 	}
	// }
	// printf("density array intitiated\n");	
	#pragma omp for
	for (i=0; i < N; i++){
		for (j=0; i < N; i++) {
			for (k=0; i < N; i++){
				den_array[i][j][k] = 0;
			}
		}
				
	}

	



	// assign values to allocated memory
	
	#pragma omp for
	for (i=0; i < M; i++) {
        den_array[(int)(particleArray[i][0]+0.5)][(int)(particleArray[i][1]+0.5)][(int)(particleArray[i][2]+0.5)] =
		den_array[(int)(particleArray[i][0]+0.5)][(int)(particleArray[i][1]+0.5)][(int)(particleArray[i][2]+0.5)] + 1;
	}
	printf("Density Array completed\n");
    // // print the 3D array
	// for (i = 0; i < I; i++)
	// {
	// 	for (j = 0; j < J; j++)
	// 	{
	// 		for (k = 0; k < K; k++)
	// 			printf("%f\n", threedArray[i][j][k]);
	//    	}
}

void center_diff(int xN, int yN, int zN, float grav_po[N][N][N], float **particleArray) {
    int i;
    float v_half, x, vx, y, vy, z, vz, X0, Y0, X1, Y1, R0, R1;
	float CM0[3], CM1[3];

    // float gx[I][J][K], gy[I][J][K], gz[I][J][K];
    // float (*g)[I][J][K];

    // for(i=1; i<xN; i++){
    //     for(j=1; j<yN-1; j++){
    //         for(k=1; k<zN-1; k++){
    //             gx[i][j][k] = (grav_po[i+1][j][k] - grav_po[i-1][j][k])/(2); // get g for each directions
    //             gy[i][j][k] = (grav_po[i][j+1][k] - grav_po[i][j-1][k])/(2);
    //             gz[i][j][k] = (grav_po[i][j][k+1] - grav_po[i][j][k-1])/(2);
    //         }
    //     }
    // }
    // printf("g force created\n");
	for (i = 0; i < 3; i ++)
	{
		CM0[i] = 0;
		CM1[i] = 0;
	}
	
	CM_finder(0, CM0, particleArray);
	CM_finder(1, CM1, particleArray);

	printf("Center of masses found\n");
    printf("%f\n", CM0[0]);
    printf("%f\n", CM0[1]);

	// #pragma omp parallel
	#pragma omp parallel for private(v_half, x, vx, y, vy, z, vz, X0, Y0, X1, Y1, R0, R1)
    for(i=0; i<M; i++){
		X0 = particleArray[i][0] - CM0[0] + 0.0001;
		Y0 = particleArray[i][1] - CM0[1] + 0.0001;
		R0 = sqrt(pow(X0,2) + pow(Y0,2)) + 0.0001;
		X1 = particleArray[i][0] - CM1[0] + 0.0001;
		Y1 = particleArray[i][1] - CM1[1] + 0.0001;
		R1 = sqrt(pow(X1,2) + pow(Y1,2)) + 0.0001;
		v_half = sqrt(pow(particleArray[i][0+3],2)+pow(particleArray[i][1+3],2)+pow(particleArray[i][2+3],2)) 
		+ 4.9*pow(10,-14)*600/(R0) + 4.9*pow(10,-14)*600/(R1) +
        (grav_po[(int)(particleArray[i][0])+1][(int)(particleArray[i][1])][(int)(particleArray[i][2])]
		- grav_po[(int)(particleArray[i][0])-1][(int)(particleArray[i][1])][(int)(particleArray[i][2])])/(4);
		x = particleArray[i][0] + v_half;
        vx = v_half + sqrt(pow(particleArray[i][0+3],2)+pow(particleArray[i][1+3],2)+pow(particleArray[i][2+3],2)) 
		+ 4.9*pow(10,-14)*600/(R0) + 4.9*pow(10,-14)*600/(R1) +
        (grav_po[(int)(particleArray[i][0])+1][(int)(particleArray[i][1])][(int)(particleArray[i][2])]
		- grav_po[(int)(particleArray[i][0])-1][(int)(particleArray[i][1])][(int)(particleArray[i][2])])/(4);
		// printf("Place 3\n");


		X0 = particleArray[i][0] - CM0[0] + 0.0001; //add 0.0001 to avoid deviding by 0 error
		Y0 = particleArray[i][1] - CM0[1] + 0.0001;
		R0 = sqrt(pow(X0,2) + pow(Y0,2)) + 0.0001;
		X1 = particleArray[i][0] - CM1[0] + 0.0001;
		Y1 = particleArray[i][1] - CM1[1] + 0.0001;
		R1 = sqrt(pow(X1,2) + pow(Y1,2)) + 0.0001;
        v_half = particleArray[i][1+3] + 4.9*pow(10,-14)*600/(R0) + 4.9*pow(10,-14)*600/(R1) +
        (grav_po[(int)(particleArray[i][0])][(int)(particleArray[i][1])+1][(int)(particleArray[i][2])] 
		- grav_po[(int)(particleArray[i][0])][(int)(particleArray[i][1])-1][(int)(particleArray[i][2])])/(4);
        y = particleArray[i][1] + v_half;
        vy = v_half + 4.9*pow(10,-14)*600/(R0) + 4.9*pow(10,-14)*600/(R1) +
        (grav_po[(int)(particleArray[i][0])][(int)(particleArray[i][1])+1][(int)(particleArray[i][2])] 
		- grav_po[(int)(particleArray[i][0])][(int)(particleArray[i][1])-1][(int)(particleArray[i][2])])/(4);
		// particleArray[i][1] = x;
		// printf("Place 4\n");

		X0 = particleArray[i][0] - CM0[0];
		Y0 = particleArray[i][1] - CM0[1];
		R0 = sqrt(pow(X0,2) + pow(Y0,2)) + 0.00001;
		// printf("Place 5\n");
		X1 = particleArray[i][0] - CM1[0];
		Y1 = particleArray[i][1] - CM1[1];
		R1 = sqrt(pow(X1,2) + pow(Y1,2)) + 0.00001;
        v_half = particleArray[i][2+3] + 4.9*pow(10,-14)*600/(R0) + 4.9*pow(10,-14)*600/(R1) +
        (grav_po[(int)(particleArray[i][0])][(int)(particleArray[i][1])][(int)(particleArray[i][2])+1] 
		- grav_po[(int)(particleArray[i][0])][(int)(particleArray[i][1])][(int)(particleArray[i][2])-1])/(4);
		// printf("Place 6\n");
		z = particleArray[i][2] + v_half;
        vz = v_half - 4.9*pow(10,-14)*600/(R0) - 4.9*pow(10,-14)*600/(R1) +
        (grav_po[(int)(particleArray[i][0])][(int)(particleArray[i][1])][(int)(particleArray[i][2])+1] 
		- grav_po[(int)(particleArray[i][0])][(int)(particleArray[i][1])][(int)(particleArray[i][2])-1])/(4);
		// particleArray[i][2+3] = v;
		particleArray[i][2] = x;
		particleArray[i][2+3] = vz;
		particleArray[i][2] = z;
		particleArray[i][1+3] = vy;
		particleArray[i][1] = y;
		particleArray[i][0+3] = vx;
		particleArray[i][0] = x;
	
	}
	printf("Updater done\n");

    // // update density array (TDB)
    // printf("density array updater initiated\n");
}

int main()
{

    
	//initialize particle array without velocity.
	int i, j, index;//, max_number, min_number, counter;
	float t, dt, X, Y, V, R;
	float **particleArray = (float **)malloc(M * sizeof(float **));
	
	t = 0.0;
	dt = 1.0;
	
	// #pragma omp for
	// for (i = 0; i < N; i++)
	// 	for (j = 0; j < N; j++)
	// 		for (k = 0; k < N; k++)
	// 			den_array[i][j][k] = 0.0;

	#pragma omp for
	for (i = 0; i < M; i++) {
		particleArray[i] = (float *)malloc(7 * sizeof(float*));

		if (particleArray[i] == NULL) {
			fprintf(stderr, "Out of memory");
			exit(0);
		}
	}

	printf("Starting to populate the particle array\n");
	// first galaxy population
	// #pragma omp parallel
	#pragma omp for
    for (i = 0; i < (int)(M*0.05/2); i++) {
        // particleArray[i][0] = 2*1.41*cos((float)(rand()%629)/100) + 96.0;
        // particleArray[i][1] = 2*1.41*sin((float)(rand()%629)/100+3.14) + 96.0;
		particleArray[i][0] = 2*1.41*((float)(rand()%2001)/1000 - 1) + 96.0;
        particleArray[i][1] = 2*1.41*((float)(rand()%2001)/1000 - 1) + 96.0;
        particleArray[i][2] = (float)(rand()%(50+1))/1000 + 128.0; // rand() % (max_number + 1 - minimum_number) + minimum_number
        X = particleArray[i][0] - 96;
        Y = particleArray[i][1] - 96;
        R = sqrt(pow(X,2) + pow(Y,2)) + 0.00001;
        V = sqrt(1190.0*G/R);
        particleArray[i][3] = Y/R*V + 0.1;
        particleArray[i][4] = X/R*V + 0.1;
        particleArray[i][5] = 0;
    }
	
	// #pragma omp parallel
	#pragma omp for
    for (index=1; index<11; index++){
        for (i = (int)(M*0.05/2+((index-1)*0.095*M/2)); i < (int)(M*0.05/2+((index)*0.095*M/2)); i++) {
            // particleArray[i][0] = (2+index)*1.41*cos((float)(rand()%629)/100) + 96.0;
            // particleArray[i][1] = (2+index)*1.41*sin((float)(rand()%629)/100+3.14) + 96.0;
			particleArray[i][0] = (2+index)*1.41*((float)(rand()%2001)/1000 - 1) + 96.0;
        	particleArray[i][1] = (2+index)*1.41*((float)(rand()%2001)/1000 - 1) + 96.0;
            particleArray[i][2] = (float)(rand()%(50+1))/1000 + 128.0;
            X = particleArray[i][0] - 96;
            Y = particleArray[i][1] - 96;
            R = sqrt(pow(X,2) + pow(Y,2) + 0.00001);
            V = sqrt(1190.0*G/R);
            particleArray[i][3] = Y/R*V + 0.1;
            particleArray[i][4] = X/R*V + 0.1;
			particleArray[i][5] = 0;
        }
    }
	
	// #pragma omp parallel
	// #pragma omp for
	for (i = 0; i < (int)(M/2); i++) {
		for (j=6;j<7;j++){
			particleArray[i][j] = 0.0; // 0.0 is indicator for Milky Way
		}
	}

	printf("Finished populating G1\n");
	// second galaxy population
	// #pragma omp parallel
	#pragma omp for
	for (i = (int)(M*0.05/2+((10)*0.095*M/2)); i < (int)(M*0.05/2+((10)*0.095*M/2))+(int)(M*0.05/2); i++) {
        // particleArray[i][0] = 2*1.41*cos((float)(rand()%629)/100)  + 160.0;
        // particleArray[i][1] = 2*1.41*sin((float)(rand()%629)/100+3.14)  + 160.0;
		particleArray[i][0] = 2*1.41*((float)(rand()%2001)/1000 - 1) + 160.0;
		particleArray[i][1] = 2*1.41*((float)(rand()%2001)/1000 - 1) + 160.0;
        particleArray[i][2] = (float)(rand()%(50+1))/1000 + 128.0;
        X = particleArray[i][0] - 96;
        Y = particleArray[i][1] - 96;
        R = sqrt(pow(X,2) + pow(Y,2) + 0.00001);
        V = sqrt(1190.0*G/R);
        particleArray[i][3] = Y/R*V - 0.1;
        particleArray[i][4] = X/R*V - 0.1;
        particleArray[i][5] = 0;
    }

	// #pragma omp parallel
	#pragma omp for
	for (index=11; index<21; index++){
        for (i = (int)(M*0.05+((index-1)*0.095*M/2)); i < (int)(M*0.05+((index)*0.095*M/2)); i++) {
            // particleArray[i][0] = (2+index-10)*1.41*cos((float)(rand()%629)/100)  + 160.0;
            // particleArray[i][1] = (2+index-10)*1.41*sin((float)(rand()%629)/100+3.14)  + 160.0;
			particleArray[i][0] = (2+index-10)*1.41*((float)(rand()%2001)/1000 - 1) + 160.0;
			particleArray[i][1] = (2+index-10)*1.41*((float)(rand()%2001)/1000 - 1) + 160.0;
            particleArray[i][2] = (float)(rand()%(150+1))/1000 + 128.0;
            X = particleArray[i][0] - 96;
            Y = particleArray[i][1] - 96;
            R = sqrt(pow(X,2) + pow(Y,2) + 0.00001);
            V = sqrt(1190*G/R);
            particleArray[i][3] = Y/R*V - 0.1;
            particleArray[i][4] = X/R*V - 0.1;
            particleArray[i][5] = 0;
        }
    }

	// #pragma omp parallel
	#pragma omp for
	for (i = (int)(M/2); i < M; i++) {
		particleArray[i][6] = 1.0; // 1.0 is indicator for Andromeda
	}
	
	printf("Finished populating G2\n");
	//create initial velocity, for each array.
	
	///Repeat until finished.
	float*** den_array = (float***)malloc(N * sizeof(float**));
	for (i = 0; i < N; i++) {
		den_array[i] = (float**)malloc(N * sizeof(float*));
        if (den_array[i] == NULL) {
			fprintf(stderr, "Out of memory");
			exit(0);
		}
        for (j = 0; j < N; j++) {
			den_array[i][j] = (float*)malloc(N * sizeof(float));
            if (den_array[i][j] == NULL) {
				fprintf(stderr, "Out of memory");
				exit(0);
			}
		}
	}
	while (t < 500)
	{
		printf("%f\n", t);
		densArray(particleArray, den_array);
		// printf("%f\n", particleArray[M/2-1][1]);
		FFT_poisson(den_array, grav_po);
		printf("FFT done\n");
		center_diff(256, 256, 256, grav_po, particleArray);
		// printf("%f\n", particleArray[M/2-1][1]);
		if (t == 0.0)
		{
			make_image(den_array, "Initial.png", "Initial density of the system");
		}

        if (t == 5.0)
		{
			make_image(den_array, "5s.png", "Density of the system after 50,000 years");
		}

        if (t == 50.0)
		{
			make_image(den_array, "5s.png", "Density of the system after 500,000 years");
		}
		
		if (t == 125.0)
		{
			make_image(den_array, "fourth.png", "Density of the system after 1,250,000 years");
		}
		
		if (t == 250.0)
		{
			make_image(den_array, "half.png", "Density of the system after 2,500,000 years");
		}
		
		if (t == 375.0)
		{
			make_image(den_array, "three_fourths.png", "Density of the system after 3,750,000 years");
		}
		
		t += dt;
	}
	
	//Fill density array with both galaxies
	//Find potential
	//update particle with potential
	
	//end.
	
	//make_image(den_array, "final.png", "Density of the system after 5,000,000 years");
	
	return 0;

}


__global__ void real2complex(hipfftComplex *c, float *a, int n)
{
    /* compute idx, idy, and idz, the location of the element in the original NxNxN array */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < n && idy < n && idz < n)
	{
		int index = idx + idy*n + idz*n*n;
		c[index].x = a[index];
		c[index].y = 0.0f;
	}
}

__global__ void complex2real_scaled(float *a, hipfftComplex *c, float scale, int n)
{
	/* compute idx and idy, the location of the element in the original NxN array */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < n && idy < n && idz < n)
	{
		int index = idx + idy*n + idz*n*n;
		a[index] = scale * c[index].x;
	}
}


__global__ void solve_poisson(hipfftComplex *c, float *k_xyz, int n)
{
	/* compute idx and idy, the location of the element in the original NxN array */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < n && idy < n && idz < n)
	{
		int index = idx + idy*n + idz*n*n;
		float scale = -(k_xyz[idx]*k_xyz[idx] + k_xyz[idy]*k_xyz[idy] + k_xyz[idz]*k_xyz[idz]) + 0.00001f;
		if (idx == 0 && idy == 0 && idz == 0) scale = 1.0f;
		scale = 1.0f / scale;
		c[index].x *= scale;
		c[index].y *= scale;
	}
}
