//Compile with: nvcc CUDAfft2.0.cu -I/home/phyd57/N_Body1/9.2/include -L/home/phyd57/N_Body1/9.2/lib64 -lcufft -o CUDAfftcu2.out

#include <hip/hip_runtime.h>

#include <hipfft/hipfft.h>

#include <stdlib.h>
#include<stdio.h>
#include <iostream>

//#include "dislin.h"

#define N 256 // N is the sidelength of the image -> N^3 pixels in entire image
#define block_size_x 2 
#define block_size_y 2
#define block_size_z 2

float den_array[N][N][N];
float grav_po[N][N][N];
float image[N][N];

__global__ void real2complex(hipfftComplex *c, float *a, int n);
__global__ void complex2real_scaled(float *a, hipfftComplex *c, float scale, int n);
__global__ void solve_poisson(hipfftComplex *c, float *kx, float *ky, float *kz, int n);


void FFT_poisson(float den_array[N][N][N], float grav_po[N][N][N], float L)
{
	int x, y, z, i;

	float *kx, *ky, *kz, *den;
	kx = (float *)malloc(sizeof(float)*N);
	ky = (float *)malloc(sizeof(float)*N);
	kz = (float *)malloc(sizeof(float)*N);
	den = (float *)malloc(sizeof(float)*N*N*N);

	float *kx_d, *ky_d, *kz_d, *den_d;
	hipfftComplex *den_complex_d;
	hipMalloc((void **)&kx_d, sizeof(float) * N);
	hipMalloc((void **)&ky_d, sizeof(float) * N);
	hipMalloc((void **)&kz_d, sizeof(float) * N);
	hipMalloc((void **)&den_d, sizeof(float) * N * N * N);
	hipMalloc((void **)&den_complex_d, sizeof(hipfftComplex) * N * N * N);

	printf("A bunch of Mallocs\n");

	#pragma omp for
	for (x = 0; x < N; x++)
		for (y = 0; y < N; y++)
			for (z = 0; z < N; z++)
				den[x + y*N + z*N*N] = den_array[x][y][z];

	printf("Den transfer\n");

	float* den_inital = (float *)malloc(sizeof(float) * N * N * N);
	for (i = 0; i < N * N; i++)
		den_inital[i] = den[i];

	for (i = 0; i < N; i++)
	{
		kx[i] = (i / N)* L * 2.0f * 3.14159f;
		ky[i] = (i / N)* L * 2.0f * 3.14159f;
		kz[i] = (i / N)* L * 2.0f * 3.14159f;
	}

	printf("ks made, Memcpy next\n");

	hipMemcpy(kx_d, kx, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(ky_d, ky, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(kz_d, kz, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(den_d, den, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	printf("Memcpy completed\n");

	hipfftHandle plan;
	hipfftPlan3d(&plan,N,N,N,HIPFFT_C2C);
	printf("Plan created\n");

	/* Compute the execution configuration, block_size_x*block_size_y*block_size_z = number of threads */
	dim3 dimBlock(block_size_x, block_size_y, block_size_z);
	dim3 dimGrid(N/dimBlock.x, N/dimBlock.y, N/dimBlock.z);
	/* Handle N not multiple of block_size_x, block_size_y, or block_size_y */
	if (N % block_size_x != 0) dimGrid.x += 1;
	if (N % block_size_y != 0) dimGrid.y += 1;
	if (N % block_size_z != 0) dimGrid.z += 1;

	real2complex<<<dimGrid, dimBlock>>>(den_complex_d, den_d, N);

	hipfftExecC2C(plan, den_complex_d, den_complex_d, HIPFFT_FORWARD);

	solve_poisson<<<dimGrid, dimBlock>>>(den_complex_d, kx_d, ky_d, kz_d, N);

	hipfftExecC2C(plan, den_complex_d, den_complex_d, HIPFFT_BACKWARD);

	float scale = 1.0f / (N*N*N);
	complex2real_scaled<<<dimGrid, dimBlock>>>(den_d, den_complex_d, scale, N);
	

	hipMemcpy(den, den_d, sizeof(float)*N*N*N, hipMemcpyDeviceToHost);

	#pragma omp for
	for (x = 0; x < N; x++)
		for (y = 0; y < N; y++)
			for (z = 0; z < N; z++)
				grav_po[x][y][z] = den[x + y*N + z*N*N];

	printf("Transfered back, next dystroy\n");

	/* Destroy plan and clean up memory on device*/
	hipFree(kx);
	hipFree(ky);
	hipFree(kz);
	hipFree(den);
	hipFree(den_inital);
	hipfftDestroy(plan);
	hipFree(den_complex_d);
	hipFree(den);
	hipFree(kx_d);
	hipFree(ky_d);
	hipFree(kz_d);

	printf("Destruction complete\n");
}

/*
void make_image(float array[N][N][N])
{
	int x, y, z;
	float Max = -500.0, Min = 500.0
	
	#pragma omp for
	for (x = 0; x < N; x++)
		for (y = 0; y < N; y++)
			image[x][y] = 0.0;

	#pragma omp for
	for (x = 0; x < N; x++)
		for (y = 0; y < N; y++)
			for (z = 0; z < N; z++)
				image[x][y] += den_array[x][y][z];

	#pragma omp for
	for (x = 0; x < N; x++)
		for (y = 0; y < N; y++)
			if (image[x][y] > Max) Max = image[x][y];
			if (image[x][y] < Min) Min = image[x][y]

	metafl("cons");
	disini();
	pagera();
	hwfont();

	titlin("Potential map", 4);
	//titlin("anthing below", 2)

	name("X [kP]", "x");
	name("Y [kP]", "y");
	name("Potential in Z", "z");

	intax()	;
	autres(800,800);
	axspos(300,1850);
	ax3len(1400,1400,1400);
	
	call labdig(4, "YZ")
	call labdig(1, "x")
	graf3(-N/2, N/2, -N/2, N/10, -N/2, N/2, -N/2, N/10, Min, Max, Min, (Max - Min)/10.0);
	crvmat((float *)XY, 800, 800, 1, 1);

	height(50);
	title();
	disfin();
}
*/

int main()
{
	int i, j, k;

	#pragma omp parallel for
	for (i = 0; i < 256; i ++)
	{
		for (j = 0; j < 256; j++)
		{
			for (k = 0; k < 256; k++)
			{
				den_array[i][j][k] = 0.0;
				grav_po[i][j][k] = 0.0;
			}
		}
	}

	den_array[128][128][128] = 500.0;

	#pragma omp parallel for
	for (i = 143; i > 113; i --)
	{
		for (j = 113; j < 143; j++)
		{
			printf("%f,", den_array[j][i][128]);
		}
		
		printf("\n");

	}

	FFT_poisson(den_array, grav_po, 128);

	printf("z = 127:\n");
	#pragma omp parallel for
	for (i = 143; i > 113; i --)
	{
		for (j = 113; j < 143; j++)
		{
			printf("%f,", grav_po[j][i][127]);
		}
		
		printf("\n");

	}
	
	printf("\n\n");
	printf("z = 128:\n");
	#pragma omp parallel for
	for (i = 143; i > 113; i --)
	{
		for (j = 113; j < 143; j++)
		{
			printf("%f,", grav_po[j][i][128]);
		}
		
		printf("\n");

	}

	printf("\n\n");
	printf("z = 129:\n");
	#pragma omp parallel for
	for (j = 143; j > 113; j --)
	{
		for (i = 113; i < 143; i++)
		{
			printf("%.1f,", grav_po[i][j][129]);
		}
		
		printf("\n");

	}

	printf("\n%f\n", grav_po[0][128][128]);
	printf("%f\n", grav_po[255][128][128]);
	printf("%f\n", grav_po[128][0][128]);
	printf("%f\n", grav_po[128][255][128]);

	//make_image(grav_po);
}

__global__ void real2complex(hipfftComplex *c, float *a, int n)
{
    /* compute idx, idy, and idz, the location of the element in the original NxNxN array */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < n && idy < n && idz < n)
	{
		int index = idx + idy*n + idz*n*n;
		c[index].x = a[index];
		c[index].y = 0.0f;
	}
}

__global__ void complex2real_scaled(float *a, hipfftComplex *c, float scale, int n)
{
	/* compute idx and idy, the location of the element in the original NxN array */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < n && idy < n && idz < n)
	{
		int index = idx + idy*n + idz*n*n;
		a[index] = scale * c[index].x;
	}
}


__global__ void solve_poisson(hipfftComplex *c, float *kx, float *ky, float *kz, int n)
{
	/* compute idx and idy, the location of the element in the original NxN array */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx < n && idy < n && idz < n)
	{
		int index = idx + idy*n + idz*n*n;
		float scale = -(kx[idx]*kx[idx] + ky[idy]*ky[idy] + kz[idz]*kz[idz]) + 0.00001f;
		if (idx == 0 && idy == 0 && idz == 0) scale = 1.0f;
		scale = 1.0f / scale;
		c[index].x *= scale;
		c[index].y *= scale;
	}
}
